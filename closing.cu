
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */
 
int M = 10; //heigth
int N = 10; //witdh

void generateImg(int witdh,int height,int *img){
    int img_size = witdh * height;
    for (int i = 0; i < img_size; i++){
        if (i < img_size/2){
            img[i]=1;
        }
        else img[i]=0;
        
    }
}

void generateClc(int *img,int *clc,int size_of_filter, int witdh, int height){
  int index = (size_of_filter-1)/2;
  for(int i =0;i<witdh;i++){
      for(int j=0;j<height;j++){
         int row = i;
         int col = j;
         int index_row = row+index;
         int index_col = col+index;
         clc[index_row+index_col*(witdh+2*index)] = img[row+col*witdh];
        }
     }      
}



__global__ void erosionImg(int *res,int *clc,int witdh,int height,int size_of_filter){
    
    int index = (size_of_filter -1) /2;
    int row = blockIdx.x * blockDim.x + threadIdx.x+ index;
    int col = blockIdx.y * blockDim.y + threadIdx.y+ index;
    int pixel = 1;
    
    for (int i = 0; i<size_of_filter; i++){
      for (int j= 0;j<size_of_filter;j++){
          if (row < witdh +index && col < height + index){
            int tmp = row + col * (witdh + index * 2);
            if (pixel > clc[row + (i - index) + (col +(j- index)) * (witdh + index *2)]){
              pixel = clc[tmp +i - index + (j- index) * (witdh+ index *2)];
            }
          }
        
      }
    }
    
    if (row < witdh +index && col < height + index){
        res[(col-index)*witdh+row-index] = pixel;
    }
    
}


void dilationCPU(int *res_cpu, int *clc, int witdh, int height, int size_of_filter){
    int index = (size_of_filter -1) /2;
    int pixel;
    for (int i = index; i<height+ index; i++){
        for (int j = index; j<witdh+index;j++){
            pixel =0;
            for (int k = 0;k<size_of_filter;k++){
                for (int l =0; l<size_of_filter;l++){
                    if (pixel < clc[j-index+k+(i-index+l)*(witdh+2*index)]){
                        pixel = clc[j-index+k+(i-index+l)*(witdh+2*index)];
                    }
                }
            }
            res_cpu[j-index + (i-index) * witdh] = pixel;
        }
    }
}

void erosionCPU(int *res_cpu, int *clc, int witdh, int height, int size_of_filter){
    int index = (size_of_filter -1) /2;
    int pixel;
    for (int i = index; i<height+ index; i++){
        for (int j = index; j<witdh+index;j++){
            pixel =1;
            for (int k = 0;k<size_of_filter;k++){
                for (int l =0; l<size_of_filter;l++){
                    if (pixel > clc[j-index+k+(i-index+l)*(witdh+2*index)]){
                        pixel = clc[j-index+k+(i-index+l)*(witdh+2*index)];
                    }
                }
            }
            res_cpu[j-index + (i-index) * witdh] = pixel;
        }
    }
}



__global__ void dilationImg(int *res,int *clc,int witdh,int height,int size_of_filter){
    
    int index = (size_of_filter -1) /2;
    int row = blockIdx.x * blockDim.x + threadIdx.x+ index;
    int col = blockIdx.y * blockDim.y + threadIdx.y+ index;
    int pixel = 0;
    
    
    for (int i = 0; i<size_of_filter; i++){
      for (int j= 0;j<size_of_filter;j++){
          if (row < witdh +index && col < height){
            int tmp = row + col * (witdh + index * 2);
            if (pixel < clc[row + (i - index) + (col +(j- index)) * (witdh + index *2)]){
              pixel = clc[tmp +i - index + (j- index) * (witdh+ index *2)];
            }
          }
        
      }
    }
    
    if (row < witdh +index && col < height){
        res[(col-index)*witdh+row-index] = pixel;
    }
}


int main()
{

  
  //Assume that the image is black and white.
  
  int witdh = N;
  int height = M;

  //filter can only be an odd number
  int size_of_filter = 3;


  //Prepare the size of the matrix who'll help us to do the calculation
  int clc_witdh = witdh + size_of_filter -1;
  int clc_height = height + size_of_filter -1;

  
  int size = witdh * height * sizeof(int);
  int clc_size = clc_witdh * clc_height * sizeof(int);
  int *img;
  int *res;
  int *res_ero_gpu;
  int *res_cpu;
  int *res_ero_cpu;
  int *clc_cpu;
  int *clc_gpu;
  hipMallocManaged(&img,size);
  hipMallocManaged(&res,size);
  hipMallocManaged(&res_cpu,size);
  hipMallocManaged(&res_ero_gpu,size);
  hipMallocManaged(&res_ero_cpu,size);
  hipMallocManaged(&clc_cpu,clc_size);
  hipMallocManaged(&clc_gpu,clc_size);
  
  //Call function to create the image;
  generateImg(witdh,height,img);
  
  printf("Image : \n");
      for (int i =0;i<height;i++){
          printf("\n");
          for (int j=0;j<witdh;j++){
              printf ("%d   ",img[i*witdh+j]);
          }
      }
  
  generateClc(img,clc_cpu,size_of_filter,witdh,height);
  generateClc(img,clc_gpu,size_of_filter,witdh,height);
  

  dim3 threads_per_block (16, 16, 1);
  dim3 number_of_blocks ((witdh / threads_per_block.x) + 1, (height / threads_per_block.y) + 1, 1);

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  dilationImg<<<number_of_blocks, threads_per_block>>>(res_ero_gpu,clc_gpu, witdh, height,size_of_filter);
  hipDeviceSynchronize();
  dilationCPU(res_ero_cpu,clc_cpu,witdh,height,size_of_filter);
  
  generateClc(res_ero_gpu,clc_gpu,size_of_filter,witdh,height);
  generateClc(res_ero_cpu,clc_cpu,size_of_filter,witdh,height);
  
  
  erosionImg<<<number_of_blocks, threads_per_block>>>(res,clc_gpu, witdh, height,size_of_filter);
  hipDeviceSynchronize();
  erosionCPU(res_cpu,clc_cpu,witdh,height,size_of_filter);
  
  printf("\n");
  
  for (int i = 0; i<N*M;i++){
     if (res[i] != res_cpu[i]){
         printf("res = %d res_cpu = %d ____ i value = %d\n",res[i],res_cpu[i],i);
     }
  }

  printf("\n\n GPU erosion: \n"); 
  for (int i =0;i<height;i++){
      printf("\n");
      for (int j=0;j<witdh;j++){
          printf ("%d   ",res[i*witdh+j]);
      }
  }
  printf("\n\n CPU erosion: \n");
  for (int i =0;i<height;i++){
      printf("\n");
      for (int j=0;j<witdh;j++){
          printf ("%d   ",res_cpu[i*witdh+j]);
      }
  }


  
  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  hipFree(img);
  hipFree(res);
  hipFree(res_ero_gpu);
  hipFree(res_cpu);
  hipFree(res_ero_cpu);
  hipFree(clc_cpu);
  hipFree(clc_gpu);
}
;
