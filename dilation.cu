

#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */
 
int M = 10; //heigth
int N = 10; //witdh

void generateImg(int witdh,int height,int *img){
    int img_size = witdh * height;
    for (int i = 0; i < img_size; i++){
        if (i < img_size/2){
            img[i]=1;
        }
        else img[i]=0;
        
    }
}

void generateClc(int *img,int *clc,int size_of_filter, int witdh, int height){
  int index = (size_of_filter-1)/2;
  for(int i =0;i<witdh;i++){
      for(int j=0;j<height;j++){
         int row = i;
         int col = j;
         int index_row = row+index;
         int index_col = col+index;
         clc[index_row+index_col*(witdh+2*index)] = img[row+col*witdh];
        }
     } 

   printf("Image : \n");
      for (int i =0;i<height;i++){
          printf("\n");
          for (int j=0;j<witdh;j++){
              printf ("%d   ",img[i*witdh+j]);
          }
      }
        
     printf("\n\nImage with adding border : \n");   
          for (int i =0;i<height+2*index;i++){
              printf("\n");
              for (int j=0;j<witdh+2*index;j++){
                  printf ("%d   ",clc[i*(witdh+2*index)+j]);
              }
          }
}


void dilationCPU(int *res_cpu, int *clc, int witdh, int height, int size_of_filter){
    int index = (size_of_filter -1) /2;
    int pixel;
    for (int i = index; i<height+ index; i++){
        for (int j = index; j<witdh+index;j++){
            pixel =0;
            for (int k = 0;k<size_of_filter;k++){
                for (int l =0; l<size_of_filter;l++){
                    if (pixel < clc[j-index+k+(i-index+l)*(witdh+2*index)]){
                        pixel = clc[j-index+k+(i-index+l)*(witdh+2*index)];
                    }
                }
            }
            res_cpu[j-index + (i-index) * witdh] = pixel;
        }
    }
}




__global__ void dilationImg(int *res,int *clc,int witdh,int height,int size_of_filter){
    
    int index = (size_of_filter -1) /2;
    int row = blockIdx.x * blockDim.x + threadIdx.x+ index;
    int col = blockIdx.y * blockDim.y + threadIdx.y+ index;
    int pixel = 0;
    
    
    for (int i = 0; i<size_of_filter; i++){
      for (int j= 0;j<size_of_filter;j++){
          if (row < witdh +index && col < height){
            int tmp = row + col * (witdh + index * 2);
            if (pixel < clc[row + (i - index) + (col +(j- index)) * (witdh + index *2)]){
              pixel = clc[tmp +i - index + (j- index) * (witdh+ index *2)];
            }
          }
        
      }
    }
    
    if (row < witdh +index && col < height){
        res[(col-index)*witdh+row-index] = pixel;
    }
}


int main()
{
  //Assume that the image is black and white.
  
  int witdh = N;
  int height = M;

  //filter can only be an odd number
  int size_of_filter = 3;


  //Prepare the size of the matrix who'll help us to do the calculation
  int clc_witdh = witdh + size_of_filter -1;
  int clc_height = height + size_of_filter -1;

  
  int size = witdh * height * sizeof(int);
  int clc_size = clc_witdh * clc_height * sizeof(int);
  int *img;
  int *res;
  int *res_cpu;
  int *clc;
  hipMallocManaged(&img,size);
  hipMallocManaged(&res,size);
  hipMallocManaged(&res_cpu,size);
  hipMallocManaged(&clc,clc_size);
  
  //Call function to create the image;
  generateImg(witdh,height,img);
  generateClc(img,clc,size_of_filter,witdh,height);
  
  
  //TIMER 
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  dim3 threads_per_block (16, 16, 1);
  dim3 number_of_blocks ((witdh / threads_per_block.x) + 1, (height / threads_per_block.y) + 1, 1);

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  dilationImg<<<number_of_blocks, threads_per_block>>>(res,clc, witdh, height,size_of_filter);
  hipDeviceSynchronize();
  
  
  //print TIMER
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("\n\nGPU execution time: %f ms\n", elapsedTime);

  //TIMER
  hipEventRecord(start);
    
  dilationCPU(res_cpu,clc,witdh,height,size_of_filter);

  //print TIMER
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("\n\nCPU execution time: %f ms\n", elapsedTime);
  
  for (int i = 0; i<N*M;i++){
     if (res[i] != res_cpu[i]){
         printf("res = %d res_cpu = %d ____ i value = %d\n",res[i],res_cpu[i],i);
     }
  }

  
  printf("\n\n GPU dilation: \n"); 
  for (int i =0;i<height;i++){
      printf("\n");
      for (int j=0;j<witdh;j++){
          printf ("%d   ",res[i*witdh+j]);
      }
  }
  printf("\n\n CPU dilation: \n");
  for (int i =0;i<height;i++){
      printf("\n");
      for (int j=0;j<witdh;j++){
          printf ("%d   ",res_cpu[i*witdh+j]);
      }
  }


  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));


  hipFree(img);
  hipFree(res);
  hipFree(res_cpu);
  hipFree(clc);
}
;
